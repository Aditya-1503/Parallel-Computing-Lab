#include <stdio.h>
#include "hip/hip_runtime.h"



__global__ void add(int *A, int *B, int *C, int m){
    int i = blockDim.x * blockIdx.x + threadIdx.x ;
    
    C[i] = A[i] + B[i];
}


void vecAdd(int *a, int *b, int m){
    int datasize = sizeof(int) * m;
    int *d_a, *d_b, *d_c;
    int *c;
    c = (int*)malloc(datasize);
    hipMalloc((void**)&d_a, datasize);
    hipMemcpy(d_a, a, datasize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_b, datasize);
    hipMemcpy(d_b, b, datasize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_c, datasize);

    //N threads 1 block
    printf("N threads 1 block\n");
    add<<<m, 1>>>(d_a, d_b, d_c, m);
    hipMemcpy(c, d_c, datasize, hipMemcpyDeviceToHost);

    for (int i=0;i <m;i++){
        printf("row %d \n" , c[i]);
    }
    

    //N blocks 1 thread each
    printf("N blocks 1 thread each\n");
    add<<<1,m>>>(d_a, d_b, d_c, m);
    hipMemcpy(c, d_c, datasize, hipMemcpyDeviceToHost);

    for (int i=0;i <m;i++){
        printf("row %d \n" , c[i]);
    }

    //256 size
    printf("256 threads per ceil(m/256) blocks\n");
    dim3 dimGrid(ceil(m/256), 1, 1);
    dim3 dimBlock(256, 1 , 1);
    add<<<dimGrid, dimBlock>>> (d_a, d_b, d_c, 256);
    hipMemcpy(c, d_c, datasize, hipMemcpyDeviceToHost);

    for (int i=0;i <m;i++){
        printf("row %d \n" , c[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}


int main(){
    int *a, *b;
    const int elements = 3;
    int datasize = elements*sizeof(int);

    a = (int*)malloc(datasize);
    b = (int*)malloc(datasize);
printf("enter elements for a:");
for(int i = 0; i < elements; i++)
{
    scanf("%d", a+i);
}

printf("enter elements for b:");
for(int i = 0; i < elements; i++)
{
   scanf("%d", b+i);
}
 printf("\n");
vecAdd( a,b,elements);

   free(a);
     free(b);
return 0; 
}
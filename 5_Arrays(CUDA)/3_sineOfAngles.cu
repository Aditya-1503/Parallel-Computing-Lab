#include <stdio.h>
#include "hip/hip_runtime.h"



__global__ void sineFunc(float *A, float *B, int m){
    int i = blockDim.x * blockIdx.x + threadIdx.x ;
    
    B[i] = sin(A[i]);
}


void sineConvert(float *a, int m){
    int datasize = sizeof(int) * m;
    float *d_a, *d_b;
    float *b;
    b = (float*)malloc(datasize);
    hipMalloc((void**)&d_a, datasize);
    hipMemcpy(d_a, a, datasize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_b, datasize);

    dim3 dimGrid(ceil(m/256.0), 1, 1);
    dim3 dimBlock(256, 1 , 1);
    sineFunc<<<dimGrid, dimBlock>>> (d_a, d_b, 256);
    hipMemcpy(b, d_b, datasize, hipMemcpyDeviceToHost);

    for (int i=0;i <m;i++){
        printf("row %f \n" , b[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
}


int main(){
    float *a;
    const int elements = 3;
    int datasize = elements*sizeof(float);

    a = (float*)malloc(datasize);
    printf("enter elements for a:");
    for(int i = 0; i < elements; i++)
    {
        scanf("%f", a+i);
    }

    sineConvert(a,elements);

    free(a);
    return 0; 
}
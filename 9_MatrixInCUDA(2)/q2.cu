#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

// Kernel for matrix transformation
__global__ void transformMatrix(int* mat, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Compute global row index

    if (row < m && row > 0) { // Ensure valid row index and skip row 0
        for (int k = 0; k < n; k++) {
            int idx = row * n + k;
            int base = mat[idx];
            int result = 1;
            for (int i = 0; i < row + 1; i++) { // Compute power using a loop
                result *= base;
            }
            mat[idx] = result;
        }
    }
}

int main() {
    int m, n;

    // Input matrix dimensions
    printf("Enter m: ");
    scanf("%d", &m);
    printf("Enter n: ");
    scanf("%d", &n);

    int* mat = (int*)malloc(m * n * sizeof(int));

    // Input matrix elements
    printf("Enter elements:\n");
    for (int i = 0; i < m; i++)
        for (int j = 0; j < n; j++)
            scanf("%d", &mat[i * n + j]);

    int* d_mat;
    hipError_t err;

    // Allocate memory on device
    err = hipMalloc((void**)&d_mat, m * n * sizeof(int));
    if (err != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString(err));
        free(mat);
        return -1;
    }

    // Copy matrix to device
    hipMemcpy(d_mat, mat, m * n * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel with multiple blocks and threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (m + threadsPerBlock - 1) / threadsPerBlock;
    transformMatrix<<<blocksPerGrid, threadsPerBlock>>>(d_mat, m, n);

    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch failed: %s\n", hipGetErrorString(err));
        hipFree(d_mat);
        free(mat);
        return -1;
    }

    // Copy transformed matrix back to host
    hipMemcpy(mat, d_mat, m * n * sizeof(int), hipMemcpyDeviceToHost);

    // Print transformed matrix
    printf("Transformed Matrix:\n");
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d ", mat[i * n + j]);
        }
        printf("\n");
    }

    // Free allocated memory
    hipFree(d_mat);
    free(mat);

    return 0;
}

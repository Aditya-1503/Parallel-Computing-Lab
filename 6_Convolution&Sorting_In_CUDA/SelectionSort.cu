#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void SelSort(int *A, int *o, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= n) return; 
    int data = A[tid], pos= 0;

    for(int i=0; i<n;i++){
        if (A[i] < data || (A[i] == data && i <tid ))
        pos++;
    }
    o[pos] = data;
}


int main(){
    int n, *arr, *out;
    printf("enter size of array: ");
    scanf("%d", &n);
    int arrSize = n * sizeof(int);

    arr = (int*)malloc(arrSize);
    out = (int*)malloc(arrSize);
    printf("Enter elements: \n");
    for(int i=0; i<n; i++)
        scanf("%d", arr + i);

    int *d_arr, *d_out;

    hipMalloc((void**)&d_arr, arrSize);
    hipMemcpy(d_arr, arr, arrSize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_out, arrSize);
    dim3 dimGrid (ceil(n/256.0), 1, 1);
    dim3 dimBlock (256, 1, 1);

    SelSort <<<dimGrid, dimBlock>>> (d_arr, d_out, n);

    hipMemcpy(out, d_out, arrSize, hipMemcpyDeviceToHost);

    for(int i=0; i<n;i++)
        printf("%d ", out[i]);

    hipFree(d_arr);
    hipFree(d_out);
    
}
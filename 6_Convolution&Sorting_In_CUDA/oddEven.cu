#include<stdio.h>
#include<hip/hip_runtime.h>


__global__ void oddEven(int *A, int n){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid % 2 != 0 && tid+1 < n){
        if (A[tid] > A[tid+1]){
            int temp = A[tid];
            A[tid] = A[tid+1];
            A[tid+1]= temp;
        }
    }
}

__global__ void evenOdd(int * A, int n){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid %2 == 0 && tid+1 < n){
        if (A[tid] > A[tid+1]){
            int temp = A[tid];
            A[tid] = A[tid+1];
            A[tid+1]= temp;
        }
    }
}

int main(){
    int n;
    printf("Enter size of array: ");
    scanf("%d", &n);
    int datasize = n * sizeof(int);
    int *arr = (int*)malloc(datasize);

    printf("Enter elements of arr: ");
    for(int i=0; i<n;i++)
        scanf("%d", arr+i);

    int *d_arr;
    hipMalloc((void**)&d_arr, datasize);
    hipMemcpy(d_arr, arr, datasize, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(n/256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);

    for(int i=0;i<n ;i++){
        oddEven <<<dimGrid, dimBlock>>>(d_arr, n);
        evenOdd <<<dimGrid, dimBlock>>>(d_arr, n);
    }

    hipMemcpy(arr, d_arr, datasize, hipMemcpyDeviceToHost);

    printf("Sorted arr: \n");
    for(int i=0; i<n;i++){
        printf("%d ", *(arr+i));
    }
    

    hipFree(d_arr);
}
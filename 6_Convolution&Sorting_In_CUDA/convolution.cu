#include <stdio.h>
#include<hip/hip_runtime.h>


__global__ void convolution(float *N, float *M, float *P, int Mask_width, int width){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    float Pvalue = 0;
    int start_point = tid - (Mask_width/2);
    for(int i=0; i<Mask_width;i++){
        if (start_point + i >= 0 && (start_point + i) < width)
            Pvalue += N[start_point+i] * M[i];
        
    }P[tid] = Pvalue;
}


int main(){
    int width, mask_width;
    float *h_a, *h_mask, *h_out;
    printf("Enter width of arr: ");
    scanf("%d", &width);
    printf("Enter width of mask: ");
    scanf("%d", &mask_width);
    int arrSize = sizeof(float) * width;
    int maskSize = sizeof(float) * mask_width;
    h_a = (float*) malloc(arrSize);
    h_mask = (float*) malloc(maskSize);
    h_out = (float *) malloc(arrSize);

    printf("Enter elements of arr: ");
    for(int i=0; i<width; i++)
        scanf("%f", h_a+i);
    
    printf("Enter elements of mask: ");
    for(int i=0; i<mask_width; i++)
        scanf("%f", h_mask+i);

    float *d_mask, *d_arr, *d_out;
    hipMalloc((void**)&d_arr, arrSize);
    hipMemcpy(d_arr, h_a, arrSize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_mask, maskSize);
    hipMemcpy(d_mask, h_mask, maskSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&d_out, arrSize);
    dim3 dimGrid(ceil(width/256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);

    convolution <<<dimGrid, dimBlock>>>(d_arr, d_mask, d_out, mask_width, width);


    hipMemcpy(h_out, d_out, arrSize, hipMemcpyDeviceToHost);

    printf("Output: \n");
    for(int i=0;i<width; i++){
        printf("%f ", h_out[i]);
    }

    hipFree(d_arr);
    hipFree(d_out);
    hipFree(d_mask);
    

}
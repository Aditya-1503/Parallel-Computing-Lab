#include <stdio.h>
#include<hip/hip_runtime.h>

#include <string.h>
#include <stdlib.h>

__global__ void count(char * sequence, char* word, int * count, int sequenceLen, int wordLen){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx + wordLen > sequenceLen) return ;


    int match = 1;
    for (int i=0;i<wordLen;i++){
        if (sequence[idx + i] != word[i]){
            match = 0;
            break;
        }
    }
    if (match) atomicAdd(count, 1);

}


int main(){
    int sequenceLen, wordLen, output=0;
    char * sequence, *word;

    printf("Enter the length of sequence: ");
    scanf("%d", &sequenceLen);
    
    printf("Enter the length of word: ");
    scanf("%d", &wordLen);

    int seqSize = (sequenceLen+1) * sizeof(char);
    int wordSize = (wordLen+1) * sizeof(char);

    sequence = (char *)malloc(seqSize);
    word = (char *)malloc(wordSize);
    
    getchar();
    
    printf("Enter the sequence: "); 
    fgets(sequence, sequenceLen+1, stdin);
    getchar();
    printf("Enter the word to search: ");
    fgets(word, wordLen+1, stdin);

    sequence[sequenceLen] = '\0';
    word[wordLen] = '\0';

    char * d_sequence, *d_word;
    int *d_out;

    hipMalloc((void**)&d_sequence, seqSize);
    hipMemcpy(d_sequence, sequence, seqSize, hipMemcpyHostToDevice);
    
    hipMalloc((void**)&d_word, wordSize);
    hipMemcpy(d_word, word, wordSize, hipMemcpyHostToDevice);
    
    hipMalloc((void**) &d_out, sizeof(int));
    hipMemcpy(d_out, &output, sizeof(int), hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(sequenceLen/256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);

    count <<<dimGrid, dimBlock>>>(d_sequence, d_word, d_out, sequenceLen, wordLen);
    hipMemcpy(&output, d_out, sizeof(int), hipMemcpyDeviceToHost);

    printf("Count: %d", output);
    

    hipFree(d_out);
    hipFree(d_sequence);
    hipFree(d_word);
    return 0;
}
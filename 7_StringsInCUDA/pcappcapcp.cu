#include<stdio.h>
#include<hip/hip_runtime.h>

#include<string.h>
#include<stdlib.h>

__global__ void prodRS(char *S, char* RS, int * pos, int n){
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    int len = n - tid;

    for(int i=0;i<len; i++)
        RS[pos[tid] + i] = S[i];
}

void positions(int * pos, int n){
    pos[0] = 0;
    for (int i=1; i<n;i++)
        pos[i] = pos[i-1] + (n-i+1);
}


int main(){
    int n, *pos;
    char *S, *RS;
    printf("Enter lenght of S:");
    scanf("%d", &n);
    int Slen = sizeof(char) * n;
    int RSlen = sizeof(char) * (n*(n+1)/2);
    
    S = (char*)malloc(Slen);
    RS = (char*)malloc(RSlen);
    pos = (int*) malloc(n*sizeof(int));


    printf("enter string: ");
    getchar();
    fgets(S, Slen+1, stdin);
    getchar();

    positions(pos, n);
    char *d_S, *d_RS;
    int *d_pos;

    hipMalloc((void**)&d_S, Slen);
    hipMemcpy(d_S, S, Slen, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_pos, n*sizeof(int));
    hipMemcpy(d_pos, pos, n*sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**)&d_RS, RSlen);
    
    dim3 dimGrid(ceil(n/256.0), 1, 1);
    dim3 dimBlock(256, 1, 1);
    prodRS<<<dimGrid, dimBlock>>>(d_S, d_RS, d_pos, n);

    hipMemcpy(RS, d_RS, RSlen, hipMemcpyDeviceToHost);

    printf("output string: %s\n", RS);
    
    hipFree(d_S);
    hipFree(d_RS);

    return 0;
}
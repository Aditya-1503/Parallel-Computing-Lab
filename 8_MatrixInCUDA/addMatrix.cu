#include<hip/hip_runtime.h>
#include<stdio.h>



__global__ void rowThread(int * A, int *B ,int *C, int n){
    int rid = threadIdx.x;
    
    if (rid < n){
        for(int i=0; i<n;i++){
            C[rid*n + i] = A[rid*n + i] + B[rid*n+i];
        }
    }

}


__global__ void colThread(int * A, int *B ,int *C, int n){
    int cid = threadIdx.x;
    
    if (cid < n){
        for(int i=0; i<n;i++){
            C[i*n + cid] = A[i*n + cid] + B[i*n + cid];
        }
    }

}


__global__ void eleThread(int * A, int *B ,int *C, int n){
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (col < n && row < n){
        int index = row * n + col;
        C[index] = A[index] + B[index];
        
    }

}



int main(){
    int A[3][3], B[3][3], C[3][3], D[3][3], E[3][3];

    printf("enter A:");
    for(int i=0;i<3;i++){
        for(int j=0;j<3; j++) scanf("%d", &A[i][j]);
    }
    printf("enter B:");
    for(int i=0;i<3;i++){
        for(int j=0;j<3; j++) scanf("%d", &B[i][j]);
    }

    int *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, sizeof(int)*9);
    hipMalloc((void**)&d_B, sizeof(int)*9);
    hipMalloc((void**)&d_C, sizeof(int)*9);

    hipMemcpy(d_A, A, 9*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, 9*sizeof(int), hipMemcpyHostToDevice);

    dim3 gridDim (1, 1);
    dim3 blockDim(3,3);
    rowThread<<<gridDim, blockDim>>>(d_A, d_B, d_C, 3);
    hipMemcpy(C, d_C, 9 * sizeof(int), hipMemcpyDeviceToHost);
    colThread<<<gridDim, blockDim>>>(d_A, d_B, d_C, 3);
    hipMemcpy(D, d_C, 9 * sizeof(int), hipMemcpyDeviceToHost);
    eleThread<<<gridDim, blockDim>>>(d_A, d_B, d_C, 3);
    hipMemcpy(E, d_C, 9 * sizeof(int), hipMemcpyDeviceToHost);

    printf("Row using one thread\n");
    for(int i=0;i<3;i++){
        for(int j=0;j<3; j++)
            printf("%d ", C[i][j]);
        printf("\n");
    }

    printf("Col using one thread\n");
    for(int i=0;i<3;i++){
        for(int j=0;j<3; j++)
            printf("%d ", D[i][j]);
        printf("\n");
    }

    printf("Element using one thread\n");
    for(int i=0;i<3;i++){
        for(int j=0;j<3; j++)
            printf("%d ", E[i][j]);
        printf("\n");
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
